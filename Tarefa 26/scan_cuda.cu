/**
* Pontifícia Universidade Católica de Minas Gerais
* Computação Paralela - Tarefa26
* Parallelized by:
* @author Ana Carolina Medeiros Gonçalves
* @author Arthur Gabriel Mathias Marques 
* @author Igor Machado Seixas
* @author Vinicius Francisco da Silva
* @version 0.01
*/

// Execution Time Serial:                           0m:00.406s      | SpeedUp

// Execution Time Parallel GPU CUDA                 0m:01.960s      | ----     
    // [CUDA memcpy HtoD]                = 469.93ms  (2 chamadas)
    // [CUDA mempcy DtoH]                = 376.38    (2 chamadas)
    // scan_cuda(double*, double*, int)  = 46.740ms
    // add_cuda(double*, double*, int)   = 15.119ms


/**
 * Implementamos a versão paralela GPU para CUDA. Observamos que esta versão possuía um tempo maior de execução.
 * Executamos o nvprof para verificação do que poderia estar acontecendo. Ao final deste arquivo segue resultado completo.
 * Observamos que ele gastou quase 1 segundo para copiar as variáveis entre a CPU e a GPU e atribuimos
 * este fato ao desempenho inferior da versão paralelizada com GPU CUDA.
 */

    
#include <hip/hip_runtime.h>
#include <stdio.h>
    #include <stdlib.h>
    
    __global__ void scan_cuda(double* a, double *s, int width) {
      // kernel scan
      int t = threadIdx.x;
      int b = blockIdx.x * blockDim.x;
      double x;
    
      // cria vetor na memória local
      __shared__ double p[1024];
    
      // carrega elementos do vetor da memória global para a local
      if(b+t < width){
          p[t] = a[b+t];
      }
        __syncthreads();
    
        for(unsigned int i=1; i < blockDim.x; i <<= 1){
            if(t >= i){ // verifica se a thread ainda participa neste passo
                x = p[t] + p[t-i];
            }
            __syncthreads();
        
    
            if(t >= i){
                p[t] = x; // copia soma em definitivo para o vetor local
            }
            __syncthreads();
        }
    
        if(b+t < width){ // copia da memória local apra a global
            a[b+t] = p[t];
        }
    
        if(t == blockDim.x-1){ // se for a última thread do bloco.
            s[blockIdx.x+1] = a[b+t]; // copia o seu valor para o vertor de saída.
        }
    } 
    
    __global__ void add_cuda(double *a, double *s, int width) {
      // kernel soma
      int t = threadIdx.x;
      int b = blockIdx.x * blockDim.x;
    
      // soma o somatório do último elemento do bloco anterior ao elemento atual
      if(b+t < width){
          a[b+t] += s[blockIdx.x];
      }
    }
    
    int main()
    {
      int width = 40000000;
      int size = width * sizeof(double);
    
      int block_size = 1024;
      int num_blocks = (width-1)/block_size+1;
      int s_size = (num_blocks * sizeof(double));  
     
      double *a = (double*) malloc (size);
      double *s = (double*) malloc (s_size);
    
      for(int i = 0; i < width; i++)
        a[i] = i;
    
      double *d_a, *d_s;
    
      // alocar vetores "a" e "s" no device
      hipMalloc((void **) &d_a, size);
      hipMalloc((void **) &d_s, s_size);
    
      // copiar vetor "a" para o device
      hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    
      // definição do número de blocos e threads (dimGrid e dimBlock)
      dim3 dimGrid(num_blocks, 1, 1);
      dim3 dimBlock(block_size, 1, 1);
    
      // chamada do kernel scan
      scan_cuda<<<dimGrid, dimBlock>>>(d_a, d_s, width);
    
      // copiar vetor "s" para o host
      hipMemcpy(s, d_s, s_size, hipMemcpyDeviceToHost);
    
      // scan no host (já implementado)
      s[0] = 0;
      for (int i = 1; i < num_blocks; i++)
        s[i] += s[i-1];
     
      // copiar vetor "s" para o device
      hipMemcpy(d_s, s, s_size, hipMemcpyHostToDevice);
    
      // chamada do kernel da soma
      add_cuda<<<dimGrid, dimBlock>>>(d_a, d_s, width);
    
      // copiar o vetor "a" para o host
      hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
    
      printf("\na[%d] = %f\n",width-1,a[width-1]);
      
      hipFree(d_a);
      hipFree(d_s);
    }
    
    /**
    ==2391== Profiling application: ./scan_cuda
    ==2391== Profiling result:
    Time(%)      Time     Calls       Avg       Min       Max  Name
    51.72%  469.39ms         2  234.69ms  452.27us  468.93ms  [CUDA memcpy HtoD]
    41.47%  376.38ms         2  188.19ms  362.28us  376.01ms  [CUDA memcpy DtoH]
    5.15%  46.740ms         1  46.740ms  46.740ms  46.740ms  scan_cuda(double*, double*, int)
    1.67%  15.119ms         1  15.119ms  15.119ms  15.119ms  add_cuda(double*, double*, int)

    ==2391== API calls:
    Time(%)      Time     Calls       Avg       Min       Max  Name
    79.37%  908.82ms         4  227.20ms  87.531us  467.78ms  cudaMemcpy
    20.50%  234.72ms         2  117.36ms  9.7090us  234.71ms  cudaMalloc
    0.06%  731.38us         2  365.69us  44.425us  686.96us  cudaFree
    0.04%  484.25us        90  5.3800us     274ns  206.65us  cuDeviceGetAttribute
    0.01%  107.11us         1  107.11us  107.11us  107.11us  cuDeviceTotalMem
    0.01%  79.656us         2  39.828us  25.777us  53.879us  cudaLaunch
    0.01%  65.786us         1  65.786us  65.786us  65.786us  cuDeviceGetName
    0.00%  10.203us         6  1.7000us     352ns  7.3800us  cudaSetupArgument
    0.00%  4.6610us         2  2.3300us     780ns  3.8810us  cudaConfigureCall
    0.00%  2.8460us         2  1.4230us     971ns  1.8750us  cuDeviceGetCount
    0.00%  1.0690us         2     534ns     495ns     574ns  cuDeviceGet
    */