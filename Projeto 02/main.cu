/**********************************************************************
 * Algoritmo de treinamento back-propagation para redes multicamadas
**********************************************************************/

/************************* BIBLIOTECAS *******************************/
#include <iostream>

#include "redeNeural.cu"

using namespace std;

/************************* DEFINICOES ********************************/
#define MAXCAM              100             // N�mero m�ximo de camadas
#define MAXNEUIN            1000            // N�mero m�ximo de neur�nios de entrada.
#define MAXNEUOUT           1000            // N�mero m�ximo de neur�nios de saida.

/****************** PROGRAMA PRINCIPAL *****************************/
int main(void)
{
  int Numero_Neuronio_Camada_Inicial;     // Número de neurônios da Camada Inicial.
  int Numero_Camadas_Escondidas;          // Número de camadas escondidas da rede.
  int Numero_Neuronio_Camadas_Escodidas; // Número de neurônios da Camada Escondida.
  int Numero_Neuronio_Saida;              // Número de neurônios da Camada de Saída.
  int i, j;

  int size = MAXNEUIN * sizeof(double);

  int block_size = 1024;
  int num_blocks = (MAXNEUIN-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));  

  double Entrada[MAXNEUIN];
  double Saida[MAXNEUOUT];

  char Continua = 'r';
  RedeNeural *R;
  RedeNeural *d_R;
  srand(time(NULL));

  while(Continua != 'n')
  {

    if(Continua == 'r')
    {
      cout << "\n\nDigite o numero de Neurônios da primeira camada da Rede Neural: ";
      cin >> Numero_Neuronio_Camada_Inicial;  

      cout << "\n\nDigite o numero de camadas internas da Rede Neural: ";
      cin >> Numero_Camadas_Escondidas;

      cout << "\n\nDigite o numero de Neurônios da(s) camada(s) internas da Rede Neural: ";
      cin >> Numero_Neuronio_Camadas_Escodidas;

      cout << "\n\nDigite o numero de Neurônios da camada de saída da Rede Neural: ";
      cin >> Numero_Neuronio_Saida;

      R = RNA_CriarRedeNeural(Numero_Camadas_Escondidas, Numero_Neuronio_Camada_Inicial, Numero_Neuronio_Camadas_Escodidas, Numero_Neuronio_Saida);
    }

    //cout << "\n\nDigite as entradas da rede:\n";
    for(j=0; j<10000; j++){

      for(i=0; i < Numero_Neuronio_Camada_Inicial; i++)
      {
        //cout << "\nEntrada " << i << " : ";
        //cin >> Entrada[i];
        Entrada[i] = i*rand()%1000;
      }
    


      //for(i=0; i <= Numero_Neuronio_Saida-1;i++)
      //{
        //cout << "\nSaida " << i << " : " << Saida[i];
      //}

      RNA_CopiarParaEntrada(R, Entrada);     /// Enviando informações para a rede neural.

      // alocar rede R no device
      hipMalloc((void **) &d_R, size);

      // copiar vetor "a" para o device
      hipMemcpy(d_R, R, size, hipMemcpyHostToDevice);

      // definição do número de blocos e threads (dimGrid e dimBlock)
      dim3 dimGrid(num_blocks, 1, 1);
      dim3 dimBlock(block_size, 1, 1);

      // chamada do kernel scan
      RNA_CalcularSaida<<<dimGrid, dimBlock>>>(d_R, MAXNEUIN);

      //RNA_CalcularSaida(R);                  /// Calculando a decisão da rede.

      // cópia dos resultados para o host
      hipMemcpy(R, d_R, size, hipMemcpyDeviceToHost);

      RNA_CopiarDaSaida(R, Saida);           /// Extraindo a decisão para vetor Saída.
		}

    cout << "\n\nContinua ? (s/n/r)\n";
    cin >> Continua;
  }


  RNA_SalvarRede(R, "RedeNeural.bin");
  R = RNA_DestruirRedeNeural(R);

  hipFree(d_R);

  return 0;
}



