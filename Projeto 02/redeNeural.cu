
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

#define TAXA_APRENDIZADO    (0.1)
#define TAXA_PESO_INICIAL   (1.0)
#define BIAS                1

//#define AtivacaoOcultas(X)         (1.0/(1.0+exp(-X)))       /// sigmoide(x)
//#define AtivacaoSaida(X)           (1.0/(1.0+exp(-X)))       /// sigmoide(x)

//#define AtivacaoOcultas(X)         tanh(X)
//#define AtivacaoSaida(X)           tanh(X)

#define AtivacaoOcultas(X)        relu(X)
#define AtivacaoSaida(X)          relu(X)

typedef struct neuronio
{
    double* Peso;
    double  Erro;
    double  Saida;

    int QuantidadeLigacoes;

}   Neuronio;

typedef struct camada
{
    Neuronio* Neuronios;

    int QuantidadeNeuronios;

}   Camada;

typedef struct redeNeural
{
    Camada  CamadaEntrada;
    Camada* CamadaEscondida;
    Camada  CamadaSaida;

    int QuantidadeEscondidas;

}   RedeNeural;


__device__ double relu(double X)
{
    if(X < 0)
    {
        return 0;
    }
    else
    {
        return X;
    }
}

double reluDx(double X)
{
    if(X < 0)
    {
        return 0;
    }
    else
    {
        return 1;
    }
}

void RNA_CopiarVetorParaCamadas(RedeNeural* Rede, double* Vetor)
{
    int j,k,l;

    j = 0;

    for(int i=0; i<Rede->QuantidadeEscondidas; i++)
    {
        for(k=0; k<Rede->CamadaEscondida[i].QuantidadeNeuronios; k++)
        {
            for(l=0; l<Rede->CamadaEscondida[i].Neuronios[k].QuantidadeLigacoes; l++)
            {
                Rede->CamadaEscondida[i].Neuronios[k].Peso[l] = Vetor[j];
                j++;
            }
        }
    }

    /////////////////////
    for(k=0; k<Rede->CamadaSaida.QuantidadeNeuronios; k++)
    {
        for(l=0; l<Rede->CamadaSaida.Neuronios[k].QuantidadeLigacoes; l++)
        {
            Rede->CamadaSaida.Neuronios[k].Peso[l] = Vetor[j];
            j++;
        }
    }
}

void RNA_CopiarParaEntrada(RedeNeural* Rede, double* VetorEntrada)
{
    int i;

    for(i=0; i<Rede->CamadaEntrada.QuantidadeNeuronios - BIAS; i++)
    {
        Rede->CamadaEntrada.Neuronios[i].Saida = VetorEntrada[i];
    }
}

int RNA_QuantidadePesos(RedeNeural* Rede)
{
    int Soma = 0;
    for(int i=0; i<Rede->QuantidadeEscondidas; i++)
    {
        for(int j=0; j<Rede->CamadaEscondida[i].QuantidadeNeuronios; j++)
        {
            Soma = Soma + Rede->CamadaEscondida[i].Neuronios[j].QuantidadeLigacoes;
        }
    }

    for(int i=0; i<Rede->CamadaSaida.QuantidadeNeuronios; i++)
    {
        Soma = Soma + Rede->CamadaSaida.Neuronios[i].QuantidadeLigacoes;
    }
    return Soma;
}

void RNA_CopiarDaSaida(RedeNeural* Rede, double* VetorSaida)
{
    int i;

    for(i=0; i<Rede->CamadaSaida.QuantidadeNeuronios; i++)
    {
        VetorSaida[i] = Rede->CamadaSaida.Neuronios[i].Saida;
    }
}

__global__ void RNA_CalcularSaida(RedeNeural* Rede, int width)
{

    // kernel scan
    int t = threadIdx.x;
    int b = blockIdx.x * blockDim.x;
    //RedeNeural x;
  
    // cria vetor na memória local
    //__shared__ RedeNeural p[1024];
    __shared__ double Somatorio;

    /*
    // carrega elementos do vetor da memória global para a local
    if(b+t < width){
        p[t] = Rede[b+t];
    }
    __syncthreads();
		*/

		//double Somatorio = 0.0;
    /// Calculando saidas entre a camada de entrada e a primeira camada escondida ///////////////////////////////////////////////////////////////////////////////
    for(unsigned int i=1; i < blockDim.x; i <<= 1)
    //for(int i=0; i<Rede->CamadaEscondida[0].QuantidadeNeuronios - BIAS; i++)
    {
        Somatorio = 0;
        for(int j=0; j<Rede->CamadaEntrada.QuantidadeNeuronios; j++)
        {
            Somatorio = Somatorio + Rede->CamadaEntrada.Neuronios[j].Saida * Rede->CamadaEscondida[0].Neuronios[b+t+i].Peso[j];
        }
        Rede->CamadaEscondida[0].Neuronios[b+t+i].Saida = AtivacaoOcultas(Somatorio);
    }
    __syncthreads();

    //////////////////////////////////////////////////////////////////////////////////
    /// Calculando saidas entre a camada escondida k e a camada escondida k-1 ///////////////////////////////////////////////////////////////////////////////
		int k;
		for(k=1; k<Rede->QuantidadeEscondidas; k++)
    {
    		//double Somatorio = 0.0;
    		Somatorio = 0.0;

    		for(unsigned int i=1; i < blockDim.x; i <<= 1)
        //for(int i=0; i<Rede->CamadaEscondida[k].QuantidadeNeuronios - BIAS; i++)
        {
            Somatorio = 0;
            for(int j=0; j<Rede->CamadaEscondida[k-1].QuantidadeNeuronios; j++)
            {
                Somatorio = Somatorio + Rede->CamadaEscondida[k-1].Neuronios[j].Saida * Rede->CamadaEscondida[k].Neuronios[b+t+i].Peso[j];
            }
            Rede->CamadaEscondida[k].Neuronios[b+t+i].Saida = AtivacaoOcultas(Somatorio);
        }
    }
		
		//double Somatorio = 0.0;
    Somatorio = 0.0;
    //////////////////////////////////////////////////////////////////////////////////
    /// Calculando saidas entre a camada de saida e a ultima camada escondida ///////////////////////////////////////////////////////////////////////////////
    for(unsigned int i=1; i < blockDim.x; i <<= 1)
    //for(int i=0; i<Rede->CamadaSaida.QuantidadeNeuronios; i++)
    {
        double Somatorio = 0;
        for(int j=0; j<Rede->CamadaEscondida[k-1].QuantidadeNeuronios; j++)
        {
            Somatorio = Somatorio + Rede->CamadaEscondida[k-1].Neuronios[j].Saida * Rede->CamadaSaida.Neuronios[b+t+i].Peso[j];
        }
        Rede->CamadaSaida.Neuronios[b+t+i].Saida = AtivacaoSaida(Somatorio);
    }
}

void RNA_CriarNeuronio(Neuronio* Neuron, int QuantidadeLigacoes)
{
    int i;

    Neuron->QuantidadeLigacoes = QuantidadeLigacoes;
    Neuron->Peso = (double*)malloc(QuantidadeLigacoes*sizeof(double));
		
    //#pragma omp for schedule(guided,2000)
    for(i=0; i<QuantidadeLigacoes; i++)
    {
        Neuron->Peso[i] = rand()%2000-1000;
    }

    Neuron->Erro = 0;
    Neuron->Saida = 1;
}

RedeNeural* RNA_CriarRedeNeural(int QuantidadeEscondidas, int QtdNeuroniosEntrada, int QtdNeuroniosEscondida, int QtdNeuroniosSaida)
{
    int i, j;

    QtdNeuroniosEntrada     = QtdNeuroniosEntrada + BIAS;
    QtdNeuroniosEscondida   = QtdNeuroniosEscondida + BIAS;

    RedeNeural* Rede = (RedeNeural*)malloc(sizeof(RedeNeural));

    Rede->CamadaEntrada.QuantidadeNeuronios = QtdNeuroniosEntrada;
    Rede->CamadaEntrada.Neuronios = (Neuronio*)malloc(QtdNeuroniosEntrada*sizeof(Neuronio));

    for(i=0; i<QtdNeuroniosEntrada; i++)
    {
        Rede->CamadaEntrada.Neuronios[i].Saida = 1.0;
    }

    Rede->QuantidadeEscondidas = QuantidadeEscondidas;
    Rede->CamadaEscondida = (Camada*)malloc(QuantidadeEscondidas*sizeof(Camada));

    for(i=0; i<QuantidadeEscondidas; i++)
    {
        Rede->CamadaEscondida[i].QuantidadeNeuronios = QtdNeuroniosEscondida;
        Rede->CamadaEscondida[i].Neuronios = (Neuronio*)malloc(QtdNeuroniosEscondida*sizeof(Neuronio));

        for(j=0; j<QtdNeuroniosEscondida; j++)
        {
            if(i == 0)
            {
                RNA_CriarNeuronio(&Rede->CamadaEscondida[i].Neuronios[j], QtdNeuroniosEntrada);
            }
            else
            {
                RNA_CriarNeuronio(&Rede->CamadaEscondida[i].Neuronios[j], QtdNeuroniosEscondida);
            }
        }
    }

    Rede->CamadaSaida.QuantidadeNeuronios = QtdNeuroniosSaida;
    Rede->CamadaSaida.Neuronios = (Neuronio*)malloc(QtdNeuroniosSaida*sizeof(Neuronio));

    for(j=0; j<QtdNeuroniosSaida; j++)
    {
        RNA_CriarNeuronio(&Rede->CamadaSaida.Neuronios[j], QtdNeuroniosEscondida);
    }

    //printf("Criada uma Rede Neural com:\n\n1 Camada de entrada com %d neuronio(s) + 1 BIAS.\n%d Camada(s) escondida(s), cada uma com %d neuronio(s) + 1 BIAS.\n1 Camada de saida com %d neuronio(s).\n", QtdNeuroniosEntrada-1, QuantidadeEscondidas, QtdNeuroniosEscondida-1, QtdNeuroniosSaida);

    return Rede;
}

RedeNeural* RNA_DestruirRedeNeural(RedeNeural* Rede)
{
    int i,j;

    free(Rede->CamadaEntrada.Neuronios);
    /////////////////////////////////////////////////////////////
    for(j=0; j<Rede->QuantidadeEscondidas; j++)
    {
        for(i=0; i<Rede->CamadaEscondida[j].QuantidadeNeuronios; i++)
        {
            free(Rede->CamadaEscondida[j].Neuronios[i].Peso);
        }
        free(Rede->CamadaEscondida[j].Neuronios);
    }
    free(Rede->CamadaEscondida);
    /////////////////////////////////////////////////////////
    for(i=0; i<Rede->CamadaSaida.QuantidadeNeuronios; i++)
    {
        free(Rede->CamadaSaida.Neuronios[i].Peso);
    }
    free(Rede->CamadaSaida.Neuronios);

    return NULL;
}

RedeNeural* RNA_CarregarRede(char* String)
{
    int i,j,k;
    FILE* f;
    RedeNeural* Temp;

    int QtdEscondida, QtdNeuroEntrada, QtdNeuroSaida, QtdNeuroEscondida;

    f = fopen(String,"rb");
    if(f != NULL)
    {
        fread(&QtdEscondida,1,sizeof(int),f);
        fread(&QtdNeuroEntrada,1,sizeof(int),f);
        fread(&QtdNeuroEscondida,1,sizeof(int),f);
        fread(&QtdNeuroSaida,1,sizeof(int),f);

        Temp = RNA_CriarRedeNeural(QtdEscondida,QtdNeuroEntrada,QtdNeuroEscondida,QtdNeuroSaida);

        for(k=0; k<Temp->QuantidadeEscondidas; k++)
        {
            for(i=0; i<Temp->CamadaEscondida[k].QuantidadeNeuronios; i++)
            {
                for(j=0; j<Temp->CamadaEscondida[k].Neuronios[i].QuantidadeLigacoes; j++)
                {
                    fread(&(Temp->CamadaEscondida[k].Neuronios[i].Peso[j]),1,8,f);
                }
            }
        }
        for(i=0; i<Temp->CamadaSaida.QuantidadeNeuronios; i++)
        {
            for(j=0; j<Temp->CamadaSaida.Neuronios[i].QuantidadeLigacoes; j++)
            {
                fread(&(Temp->CamadaSaida.Neuronios[i].Peso[j]),1,8,f);
            }
        }

        fclose(f);
        return Temp;
    }
}

void RNA_SalvarRede(RedeNeural* Temp, const char* String)
{
    int i,j,k;
    FILE* f;

    f = fopen(String,"wb");
    if(f != NULL)
    {
        fwrite(&Temp->QuantidadeEscondidas,1,sizeof(int),f);
        fwrite(&Temp->CamadaEntrada.QuantidadeNeuronios,1,sizeof(int),f);
        fwrite(&Temp->CamadaEscondida[0].QuantidadeNeuronios,1,sizeof(int),f);
        fwrite(&Temp->CamadaSaida.QuantidadeNeuronios,1,sizeof(int),f);

        for(k=0; k<Temp->QuantidadeEscondidas; k++)
        {
            for(i=0; i<Temp->CamadaEscondida[k].QuantidadeNeuronios; i++)
            {
                for(j=0; j<Temp->CamadaEscondida[k].Neuronios[i].QuantidadeLigacoes; j++)
                {
                    fwrite(&Temp->CamadaEscondida[k].Neuronios[i].Peso[j],1,8,f);
                }
            }
        }

        for(i=0; i<Temp->CamadaSaida.QuantidadeNeuronios; i++)
        {
            for(j=0; j<Temp->CamadaSaida.Neuronios[i].QuantidadeLigacoes; j++)
            {
                fwrite(&Temp->CamadaSaida.Neuronios[i].Peso[j],1,8,f);
            }
        }

        fclose(f);
    }
}
