/**
* Pontifícia Universidade Católica de Minas Gerais
* Computação Paralela - Tarefa23
* Parallelized by:
* @author Ana Carolina Medeiros Gonçalves
* @author Arthur Gabriel Mathias Marques 
* @author Igor Machado Seixas
* @author Vinicius Francisco da Silva
* @version 0.01
*/

// Execution Time Serial:                                       1m:15.600s      | SpeedUp
// Execution Time Parallel CPU     			                        0m:17.415s      | 4.34			
// Execution Time Parallel GPU Distribute	                      1m:50.018s      | ----
// Execution Time Parallel GPU Distribute Parrallel For         0m:20.861s      | 3.62
    // Warps Launched             = 72.
    // Warp Execution Efficiency  = 100%
// Execution Time Parallel GPU Distribute Parrallel For Simd    0m:04.517s      | 16.73     
    // Warps Launched             = 72.
    // Warp Execution Efficiency  = 86.81%
//####NOVO####
// Execution Time Parallel GPU CUDA                             0m:02.041s      | 37.04     **BEST SPEEDUP**
    // Warps Launched             = 125000.
    // Warp Execution Efficiency  = 100.00%



/**
 * Foi observado e executado o código paralela para multicore, 
 * e paralela para GPU com as diretivas "distribute", "distribute parallel for" 
 * e "distribute parallel for simd" e usando CUDA. Verificamos na versão multicore um SpeedUp de 4.34. 
 * Para a primeira experiêcia usando somente o "distribute" não gerou SpeedUp, 
 * devido a isto e a demora para o teste com o  nvprof optamos em não verificar os Warps Launched e 
 * Warp Execution Efficiency.
 * 
 * Para a versão usando "parallel for" obtivemos um SpeedUp de 3.62 ainda inferior ao CPU. Embora verificado que
 * a eficiencia no Warp Execution é maior que usando o SIMD, o melhor SpeedUp ficou usando as diretivas 
 * "distribute parallel for simd". Obtemos um SpeedUp de 16.73  muito superior ao CPU.
 * 
 * ####NOVO####
 * Por último foi implementado a versão usando o CUDA. Verificamos que, como a linguagem CUDA é escrita pela Nvidia
 * e de uso específico para GPU, o SpeedUp e eficiência aumentaram muito. Alcançando 37.04 de SpeedUp com uma
 * eficiência de 100%.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mm_cuda(double* a, double* b, double* c, int width) 
{
	//#pragma omp parallel for simd schedule(static, 100)
	//#pragma omp target map(tofrom:c[0:size]) map(to:a[0:size],b[0:size])
	//#pragma omp teams distribute parallel for simd
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	int j = blockIdx.y*blockDim.y+threadIdx.y;

	if(i<width){
		if(j<width){
      double sum = 0;
      for (int k = 0; k < width; k++) {
				double x = a[i * width + k];
				double y = b[k * width + j];
				sum += x * y;
      }
  	c[i * width + j] = sum;
		}
	}
}

int main()
{
  int width = 2000;
  double *a = (double*) malloc (width * width * sizeof(double));
  double *b = (double*) malloc (width * width * sizeof(double));
  double *c = (double*) malloc (width * width * sizeof(double));

  for(int i = 0; i < width; i++) {
    for(int j = 0; j < width; j++) {
      a[i*width+j] = i;
      b[i*width+j] = j;
      c[i*width+j] = 0;
    }
  }
	
	int size = width*width*sizeof(double);
	double *d_a, *d_b, *d_c;

	hipMalloc((void**) &d_a, size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	
	hipMalloc((void**) &d_b, size);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	hipMalloc((void**) &d_c, size);

	int block_size = 8;
	dim3 dimGrid((width-1)/block_size+1, (width-1)/block_size+1, 1);
	dim3 dimBlock(block_size, block_size, 1);

	mm_cuda<<<dimGrid,dimBlock>>>(d_a, d_b, d_c, width);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  	//mm(a,b,c,width);

	//for(int i = 0; i < width; i++) {
    	//for(int j = 0; j < width; j++) {
      		//printf("\n c[%d][%d] = %f",i,j,c[i*width+j]);
    	//}
   	//}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

}
